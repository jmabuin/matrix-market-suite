/**
  * Copyright 2019 José Manuel Abuín Mosquera <josemanuel.abuin@usc.es>
  *
  * This file is part of Matrix Market Suite.
  *
  * Matrix Market Suite is free software: you can redistribute it and/or modify
  * it under the terms of the GNU General Public License as published by
  * the Free Software Foundation, either version 3 of the License, or
  * (at your option) any later version.
  *
  * Matrix Market Suite is distributed in the hope that it will be useful,
  * but WITHOUT ANY WARRANTY; without even the implied warranty of
  * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
  * GNU General Public License for more details.
  *
  * You should have received a copy of the GNU General Public License
  * along with Matrix Market Suite. If not, see <http://www.gnu.org/licenses/>.
  */

#include <stdio.h>
#include <string.h>

extern "C" {
    #include "../utils/utils.h"
}
#include "operations/DMxV-CUDA.h"

#include "solvers/ConjugateGradient-CUDA.h"

#ifndef PACKAGE_VERSION
#define PACKAGE_VERSION "0.3.0"
#endif

typedef enum {TYPEOP, TYPESOLVER} commandtype;

struct st_command {
    const char * name;
    const char * description;
    commandtype type;
    int (*function)(int, char*[]);
};

#define NEW_OPERATION(name, description, function_name) {#name, description, TYPEOP, function_name}
#define NEW_SOLVER(name, description, function_name) {#name, description, TYPESOLVER, function_name}

const struct st_command commands[] = {
        NEW_OPERATION(DMxV,			"Dense matrix dot vector operation", DMxVCUDA),
        NEW_SOLVER(ConjugateGradient,		"Solves a system by using the conjugate gradient method", ConjugateGradientCUDA),
};

#define SHOW_COMMANDS(mytype) \
	for (i = 0, imax = sizeof(commands) / sizeof(com); i < imax; i++) { \
		com = commands[i]; \
		if (com.type != mytype) continue; \
		fprintf(stderr, "   %-34s  %s\n", com.name, com.description); \
	}

static int usage()
{
    fprintf(stderr, "\n");
    fprintf(stderr, "Program: MM-Suite (perform operations to operate over matrix market files using CUDA)\n");
    fprintf(stderr, "Version: %s\n", PACKAGE_VERSION);
    fprintf(stderr, "Contact: José M. Abuín <josemanuel.abuin@usc.es>\n\n");
    fprintf(stderr, "Usage:   MM-Suite <command> [options]\n\n");
    fprintf(stderr, "Available commands:\n");

    size_t i, imax;
    struct st_command com;

    fprintf(stderr, "\nBasic operations:\n");
    SHOW_COMMANDS(TYPEOP);

    fprintf(stderr, "\nSolvers:\n");
    SHOW_COMMANDS(TYPESOLVER);

    fprintf(stderr, "\n");
    return 1;
}

void devicenfo(void)
{
    struct hipDeviceProp_t capabilities;

    hipGetDeviceProperties (&capabilities, 0);

    printf("->CUDA Platform & Capabilities\n");
    printf("Name: %s\n", capabilities.name);
    printf("totalGlobalMem: %.2f MB\n", capabilities.totalGlobalMem/1024.0f/1024.0f);
    printf("sharedMemPerBlock: %.2f KB\n", capabilities.sharedMemPerBlock/1024.0f);
    printf("regsPerBlock (32 bits): %d\n", capabilities.regsPerBlock);
    printf("warpSize: %d\n", capabilities.warpSize);
    printf("memPitch: %.2f KB\n", capabilities.memPitch/1024.0f);
    printf("maxThreadsPerBlock: %d\n", capabilities.maxThreadsPerBlock);
    printf("maxThreadsDim: %d x %d x %d\n", capabilities.maxThreadsDim[0],
           capabilities.maxThreadsDim[1], capabilities.maxThreadsDim[2]);
    printf("maxGridSize: %d x %d\n", capabilities.maxGridSize[0],
           capabilities.maxGridSize[1]);
    printf("totalConstMem: %.2f KB\n", capabilities.totalConstMem/1024.0f);
    printf("major.minor: %d.%d\n", capabilities.major, capabilities.minor);
    printf("clockRate: %.2f MHz\n", capabilities.clockRate/1024.0f);
    printf("textureAlignment: %d\n", capabilities.textureAlignment);
    printf("deviceOverlap: %d\n", capabilities.deviceOverlap);
    printf("multiProcessorCount: %d\n", capabilities.multiProcessorCount);
}

int main(int argc, char *argv[]) {

    int i, ret;
    double t_real;

    struct st_command com;

    size_t imax;

    t_real = realtime();


    fprintf(stderr, "MM-Suite\tVN:%s\tCL:%s", PACKAGE_VERSION, argv[0]);
    fprintf(stderr, "\n");
    devicenfo();
    fprintf(stderr, "\n");

    if (argc < 2) return usage();


    for (i = 0, imax = sizeof(commands) / sizeof(com); i < imax; i++) {
        com = commands[i];
        if (strcmp(com.name, argv[1]) == 0) {
            ret = com.function(argc-1, argv+1);
            break;
        }
    }

    if (i == imax) {
        fprintf(stderr, "[%s] unrecognized command '%s'\n",__func__, argv[1]);
        return 1;
    }


    if (ret == 0) {
        fprintf(stderr, "[%s] ERROR!\n", __func__);
        fprintf(stderr, "[%s] Version: %s\n", __func__, PACKAGE_VERSION);
        fprintf(stderr, "[%s] CMD:", __func__);
        for (i = 0; i < argc; ++i)
            fprintf(stderr, " %s", argv[i]);

    }


    fprintf(stderr, "\n[%s] Real time: %.6f sec; CPU: %.6f sec; User: %.6f sec; Sys: %6f sec\n", __func__, realtime() - t_real, cputime(),usertime(),systemtime());

    if(ret == 1){
        ret = 0;
    }
    else if(ret == 0){
        ret = 1;
    }

    return ret;
}
