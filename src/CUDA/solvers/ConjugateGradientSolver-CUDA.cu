#include "hip/hip_runtime.h"
/**
  * Copyright 2019 José Manuel Abuín Mosquera <josemanuel.abuin@usc.es>
  *
  * This file is part of Matrix Market Suite.
  *
  * Matrix Market Suite is free software: you can redistribute it and/or modify
  * it under the terms of the GNU General Public License as published by
  * the Free Software Foundation, either version 3 of the License, or
  * (at your option) any later version.
  *
  * Matrix Market Suite is distributed in the hope that it will be useful,
  * but WITHOUT ANY WARRANTY; without even the implied warranty of
  * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
  * GNU General Public License for more details.
  *
  * You should have received a copy of the GNU General Public License
  * along with Matrix Market Suite. If not, see <http://www.gnu.org/licenses/>.
  */

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <>
#include <hipblas.h>
#include "ConjugateGradientSolver-CUDA.h"

int ConjugateGradientSolverCUDA(unsigned long *II, unsigned long *J, double *A, unsigned long M, unsigned long N, unsigned long long nz, double *b, unsigned long M_Vector, unsigned long N_Vector, unsigned long long nz_vector, double *x, int iterationNumber) {

    //A*x=b

    hipError_t  cudaStat;  //  hipMalloc  status
    hipblasStatus_t  stat;   //  CUBLAS  functions  status
    hipblasHandle_t  handle;

    double *d_A;
    double *d_x;
    double *d_b;
    double *Ap; //=(double *) malloc(nz_vector * sizeof(double));
    double *r; //=(double *) malloc(nz_vector * sizeof(double));
    double *p; //=(double *) malloc(nz_vector * sizeof(double));
    //double *x=(double *) calloc(nz_vector,sizeof(double));

    double			alpha = 1.0;
    double			beta = 0.0;

    // ================= Reserve memory and copy data for matrix A on device =================
    cudaStat = hipMalloc ((void **)&d_A , nz * sizeof(double));     // device

    if (cudaStat != hipSuccess) {
        fprintf(stderr, "[%s] Error in hipMalloc for matrix values\n",__func__);
        return 0;
    }

    cudaStat = hipMemcpy( d_A, A, nz * sizeof(double), hipMemcpyHostToDevice );
    if (cudaStat != hipSuccess) {
        fprintf(stderr, "[%s] Error in hipMemcpy for matrix values\n",__func__);
        return 0;

    }
    // =======================================================================================

    // ================= Reserve memory and copy data for vector b on device =================
    cudaStat = hipMalloc ((void **)&d_b , nz_vector * sizeof(double));     // device

    if (cudaStat != hipSuccess) {
        fprintf(stderr, "[%s] Error in hipMalloc for vector values\n",__func__);
        return 0;
    }

    cudaStat = hipMemcpy( d_b, b, nz_vector* sizeof(double), hipMemcpyHostToDevice );
    if (cudaStat != hipSuccess) {
        fprintf(stderr, "[%s] Error in hipblasSetVector for matrix values\n",__func__);
        return 0;

    }
    // =======================================================================================

    // ================= Reserve memory and copy data for vector x on device =================
    cudaStat = hipMalloc ((void **)&d_x , nz_vector * sizeof(double));     // device

    if (cudaStat != hipSuccess) {
        fprintf(stderr, "[%s] Error in hipMalloc for vector values\n",__func__);
        return 0;
    }

    cudaStat = hipMemcpy( d_x, x, nz_vector* sizeof(double), hipMemcpyHostToDevice );
    if (cudaStat != hipSuccess) {
        fprintf(stderr, "[%s] Error in hipblasSetVector for matrix values\n",__func__);
        return 0;

    }
    // =======================================================================================

    // ================= Reserve memory for vector Ap on device =================
    cudaStat = hipMalloc ((void **)&Ap , nz_vector * sizeof(double));     // device

    if (cudaStat != hipSuccess) {
        fprintf(stderr, "[%s] Error in hipMalloc for vector values\n",__func__);
        return 0;
    }
    // ==========================================================================

    // ================= Reserve memory for vector r on device =================
    cudaStat = hipMalloc ((void **)&r , nz_vector * sizeof(double));     // device

    if (cudaStat != hipSuccess) {
        fprintf(stderr, "[%s] Error in hipMalloc for vector values\n",__func__);
        return 0;
    }
    // ==========================================================================

    // ================= Reserve memory for vector p on device =================
    cudaStat = hipMalloc ((void **)&p , nz_vector * sizeof(double));     // device

    if (cudaStat != hipSuccess) {
        fprintf(stderr, "[%s] Error in hipMalloc for vector values\n",__func__);
        return 0;
    }
    // ==========================================================================

    stat = hipblasCreate (& handle );   //  initialize  CUBLAS  context

    if (stat != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "[%s] Error in hipblasCreate\n",__func__);
        return 0;

    }

    //r = b-A*x
    //If we take x=0 the init multiplication is avoided and r=b

    //memcpy(r, b, N*sizeof(double));
    cudaStat = hipMemcpy( r, b, nz_vector* sizeof(double), hipMemcpyHostToDevice );
    if (cudaStat != hipSuccess) {
        fprintf(stderr, "[%s] Error in hipblasSetVector for vector values\n",__func__);
        return 0;

    }

    //p=r

    //memcpy(p, r, N*sizeof(double));
    cudaStat = hipMemcpy( p, b, nz_vector* sizeof(double), hipMemcpyHostToDevice );
    if (cudaStat != hipSuccess) {
        fprintf(stderr, "[%s] Error in hipblasSetVector for vector values\n",__func__);
        return 0;

    }

    //rsold = r*r
    double rsold;// = cblas_ddot(N,r,1,r,1);

    hipblasDdot(handle, nz_vector, r, 1, r, 1, &rsold);

    int stop = 0;

    double alphaCG = 0.0;

    double rsnew = 0.0;
    unsigned long k = 0;

    unsigned long maxIterations = M*2;

    if(iterationNumber != 0 ){
        maxIterations = iterationNumber;
    }

    //int i  = 0;

    double tmp_result;

    while(!stop){

        //Ap=A*p
        // for(i=0; i<M; i++){
        //	Ap[i] = 0.0;
        // }
        //cblas_dgemv(CblasRowMajor, CblasNoTrans, M,N , 1.0, A, N, p, 1, 0.0, Ap, 1);
        stat = hipblasDgemv(handle, HIPBLAS_OP_N, M, N, &alpha, d_A, M, p, 1, &beta, Ap, 1);

        //alphaCG=rsold/(p'*Ap)
        //alphaCG = rsold/cblas_ddot(N,p,1,Ap,1);
        hipblasDdot(handle, N, p, 1, Ap, 1, &tmp_result);
        alphaCG = rsold/tmp_result;

        //x=x+alphaCG*p
        hipblasDaxpy( handle, N, &alphaCG, p, 1, d_x, 1);
        //cblas_daxpy(N,alphaCG,p,1,x,1);

        //r=r-alphaCG*Ap
        //cblas_daxpy(N,-alphaCG,Ap,1,r,1);
        tmp_result = alphaCG * -1;
        hipblasDaxpy( handle, N, &tmp_result, Ap, 1, r, 1);

        //rsnew = r'*r
        //rsnew = cblas_ddot(N,r,1,r,1);
        hipblasDdot(handle, N, r, 1, r, 1, &rsnew);

        if((sqrt(rsnew)<=EPSILON)||(k == maxIterations)){
            stop = 1;
        }

        //p=r+rsnew/rsold*p
        //cblas_dscal(N, rsnew/rsold, p, 1);
        tmp_result = (rsnew/rsold);
        hipblasDscal(handle, N, &tmp_result, p, 1);
        //cblas_daxpy(N,1.0,r,1,p,1);
        hipblasDaxpy( handle, N, &alpha, r, 1, p, 1);

        rsold = rsnew;

        k++;
    }

    //memcpy(b, x, N*sizeof(double));
    cudaStat = hipMemcpy( x, d_x, nz_vector* sizeof(double), hipMemcpyDeviceToHost );
    if (cudaStat != hipSuccess) {
        fprintf(stderr, "[%s] Error copying results back to host\n",__func__);
        return 0;

    }


    hipFree(d_x);                             // free  device  memory
    hipFree(d_A);                             // free  device  memory
    hipFree(d_b);                             // free  device  memory
    hipFree(Ap);                             // free  device  memory
    hipFree(r);                             // free  device  memory
    hipFree(p);                             // free  device  memory
    hipblasDestroy(handle);

    fprintf(stderr, "[%s] Number of iterations %lu\n",__func__,k);

    return 1;
}
