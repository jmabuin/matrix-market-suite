/**
  * Copyright 2016 José Manuel Abuín Mosquera <josemanuel.abuin@usc.es>
  *
  * This file is part of Matrix Market Suite.
  *
  * Matrix Market Suite is free software: you can redistribute it and/or modify
  * it under the terms of the GNU General Public License as published by
  * the Free Software Foundation, either version 3 of the License, or
  * (at your option) any later version.
  *
  * Matrix Market Suite is distributed in the hope that it will be useful,
  * but WITHOUT ANY WARRANTY; without even the implied warranty of
  * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
  * GNU General Public License for more details.
  *
  * You should have received a copy of the GNU General Public License
  * along with Matrix Market Suite. If not, see <http://www.gnu.org/licenses/>.
  */

#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <>
#include <hipblas.h>

#include "DMxV-CUDA.h"

void usageDMxV(){

    fprintf(stderr, "\n");
    fprintf(stderr, "Usage: MM-Suite DMxV [options] <input-matrix> <input-vector>\n");
    fprintf(stderr, "\nInput/output options:\n\n");
    fprintf(stderr, "       -o STR        Output file name. Default: stdout\n");
    fprintf(stderr, "       -r            Input format is row per line. Default: False\n");
    fprintf(stderr, "\nParameters options:\n\n");
    fprintf(stderr, "       -a DOUBLE     Alpha. Default: 1.0\n");
    fprintf(stderr, "       -b DOUBLE     Beta. Default: 0.0\n");
    fprintf(stderr, "\nPerformance options:\n\n");
    fprintf(stderr, "       -t INT        Number of threads to use in OpenBLAS. Default: 1\n");
    fprintf(stderr, "\n");

}

int DMxVCUDA(int argc, char *argv[]) {

    int 			ret_code = 1;
    int 			option;

    unsigned long 		*II;
    unsigned long 		*J;
    double 			*values;
    double          *d_values;

    unsigned long 		M;
    unsigned long 		N;
    unsigned long long 	nz;


    double 			*vectorValues;
    double 			*d_vectorValues;
    unsigned long 		M_Vector;
    unsigned long 		N_Vector;
    unsigned long long 	nz_vector;

    char			*outputFileName = NULL;

    char			*inputMatrixFile = NULL;
    char			*inputVectorFile = NULL;
    char			*outputVectorFile = NULL;

    int			inputFormatRow = 0;
    int			basicOps = 0;
    int			numThreads = 1;

    double			alpha = 1.0;
    double			beta = 0.0;

    hipError_t  cudaStat;  //  hipMalloc  status
    hipblasStatus_t  stat;   //  CUBLAS  functions  status
    hipblasHandle_t  handle;

    while ((option = getopt(argc, argv,"ero:b:a:t:")) >= 0) {
        switch (option) {
            case 'o' :
                //free(outputFileName);

                outputFileName = (char *) malloc(sizeof(char)*strlen(optarg)+1);
                strcpy(outputFileName,optarg);

                break;

            case 'r':
                inputFormatRow = 1;
                break;

            case 'e':
                basicOps = 1;
                break;

            case 'b':
                beta = atof(optarg);
                break;

            case 'a':
                alpha = atof(optarg);
                break;

            case 't':
                numThreads = atoi(optarg);
                break;

            default: break;
        }

    }

    if ((optind + 3 != argc) && (optind + 2 != argc)) {
        usageDMxV();
        return 0;
    }

    //openblas_set_num_threads(numThreads);

    if(optind + 3 == argc) { //We have an output vector

        outputVectorFile = (char *)malloc(sizeof(char)*strlen(argv[optind+2])+1);
        strcpy(outputVectorFile,argv[optind+2]);
    }

    if(outputFileName == NULL) {
        outputFileName = (char *) malloc(sizeof(char)*7);
        sprintf(outputFileName,"stdout");
    }

    inputMatrixFile = (char *)malloc(sizeof(char)*strlen(argv[optind])+1);

    if(inputMatrixFile == NULL) {
        fprintf(stderr, "[%s] Error reserving memory for input matrix file name\n",__func__);
        return 0;
    }

    inputVectorFile = (char *)malloc(sizeof(char)*strlen(argv[optind+1])+1);

    if(inputVectorFile == NULL) {
        fprintf(stderr, "[%s] Error reserving memory for input vector file name\n",__func__);
        return 0;
    }

    strcpy(inputMatrixFile,argv[optind]);
    strcpy(inputVectorFile,argv[optind+1]);

    //Read matrix
    if(inputFormatRow){

        if(!readDenseCoordinateMatrixRowLine(inputMatrixFile,&II,&J,&values,&M,&N,&nz)){
            usageDMxV();
            fprintf(stderr, "[%s] Can not read Matrix\n",__func__);
            return 0;
        }

    }
    else {
        if(!readDenseCoordinateMatrix(inputMatrixFile,&II,&J,&values,&M,&N,&nz)){
            usageDMxV();
            fprintf(stderr, "[%s] Can not read Matrix\n",__func__);
            return 0;
        }
    }

    //Read vector
    if(!readDenseVector(inputVectorFile, &vectorValues,&M_Vector,&N_Vector,&nz_vector)){
        usageDMxV();
        fprintf(stderr, "[%s] Can not read Vector\n",__func__);
        return 0;
    }

    cudaStat = hipMalloc ((void **)&d_vectorValues , nz_vector * sizeof(double));     // device

    if (cudaStat != hipSuccess) {
        fprintf(stderr, "[%s] Error in hipMalloc for vector values\n",__func__);
        return 0;
    }

    cudaStat = hipMalloc ((void **)&d_values , nz * sizeof (double));     // device

    if (cudaStat != hipSuccess) {
        fprintf(stderr, "[%s] Error in hipMalloc for matrix values\n",__func__);
        return 0;
    }

    stat = hipblasCreate (& handle );   //  initialize  CUBLAS  context

    if (stat != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "[%s] Error in hipblasCreate\n",__func__);
        return 0;

    }

    //stat = hipblasSetVector(nz_vector,sizeof (*vectorValues),vectorValues,1,d_vectorValues ,1);// cp x->d_x
    cudaStat = hipMemcpy( d_vectorValues, vectorValues, nz_vector* sizeof(double), hipMemcpyHostToDevice );
    if (cudaStat != hipSuccess) {
        fprintf(stderr, "[%s] Error in hipblasSetVector for vector values\n",__func__);
        return 0;

    }

    //stat = hipblasSetVector(nz,sizeof (*values),values,1,d_values ,1);// cp x->d_x
    cudaStat = hipMemcpy( d_values, values, nz* sizeof(double), hipMemcpyHostToDevice );
    if (cudaStat != hipSuccess) {
        fprintf(stderr, "[%s] Error in hipblasSetVector for matrix values\n",__func__);
        return 0;

    }


    double *result=(double *) malloc(nz_vector * sizeof(double));
    double *d_result;

    //Read output vector if any
    if(outputVectorFile != NULL) {
        if(!readDenseVector(outputVectorFile, &result,&M_Vector,&N_Vector,&nz_vector)){
            usageDMxV();
            fprintf(stderr, "[%s] Can not read Vector %s\n",__func__, outputVectorFile);
            return 0;
        }
    }

    cudaStat = hipMalloc ((void **)&d_result , nz_vector * sizeof (double));     // device
    if (cudaStat != hipSuccess) {
        fprintf(stderr, "[%s] Error in hipMalloc for result values\n",__func__);
        return 0;
    }

    //stat = hipblasSetVector(nz_vector,sizeof (*result),result,1,d_result ,1);// cp x->d_x
    cudaStat = hipMemcpy( d_result, result, nz_vector* sizeof(double), hipMemcpyHostToDevice );
    if (cudaStat != hipSuccess) {
        fprintf(stderr, "[%s] Error in hipblasSetVector for result vector values\n",__func__);
        return 0;

    }

    double t_real = realtime();

    /*
    hipblasStatus_t hipblasDgemv(hipblasHandle_t handle, hipblasOperation_t trans,
                           int m, int n,
                           const double          *alpha,
                           const double          *A, int lda,
                           const double          *x, int incx,
                           const double          *beta,
                           double          *y, int incy)
                 */


    stat = hipblasDgemv(handle, HIPBLAS_OP_N, M, N, &alpha, d_values, M, d_vectorValues, 1, &beta, d_result, 1);

    // Copy array back to host
    hipMemcpy( result, d_result, nz_vector* sizeof(double), hipMemcpyDeviceToHost );

    hipFree(d_result);                             // free  device  memory
    hipFree(d_values);                             // free  device  memory
    hipFree(d_vectorValues);                             // free  device  memory
    hipblasDestroy(handle);

    fprintf(stderr, "\n[%s] Time spent in hipblasDgemv: %.6f sec\n", __func__, realtime() - t_real);

    writeDenseVector(outputFileName, result,M_Vector,N_Vector,nz_vector);


    return ret_code;
}

